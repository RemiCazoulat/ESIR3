#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <unistd.h>
#include <assert.h>
#include <time.h>
#include "bits_cuda_simpler.cu"

__host__ bits_t* bits_from_long(size_t n, unsigned long value){
    if(n == 0) return NULL;
    bits_t* bits = bits_allocate(n);
    printf("[DEBUG] n_bytes : %zu\n", bits->nbytes);
    //for(long i = (long)bits->nbytes - 1; i >= 0; i --) {
    for(long i = 0; i < (long)bits->nbytes - 1; i ++) {
        char byte = (char)value;
        bits->byte[i] = byte;
        value = value >> 8;
    }
    return bits;
}

__host__ unsigned long bits_to_long(bits_t* bits){
    unsigned long final_long = 0;
    for(int i = 1; i < bits->nbytes; i ++) {
        unsigned long current_long = (long)bits->byte[i] << (bits->nbytes - 1 - i) * 8;
        final_long += current_long;
    }
    // shifting last byte 2 times to get rid of ignored information
    unsigned char last_char = (unsigned char)bits->byte[0];
    last_char = (unsigned char)(last_char << bits->ignore);
    last_char = (unsigned char)(last_char >> bits->ignore);
    unsigned long last_long = (unsigned long)last_char << (bits->nbytes - 1) * 8;
    final_long += last_long;
    return final_long;
}

// bits_from_long, but on GPU
__global__ void generate_ids(unsigned char* bits, size_t nbytes, unsigned long* debug) {
    unsigned long id = (blockIdx.x * blockDim.x) + threadIdx.x;
    debug[id] = id;
    unsigned long value = id;
    for(long i = nbytes - 1; i >= 0; i --) {
        unsigned char byte = (unsigned char)value;
        bits[i + id * nbytes] = byte;
        value = value >> 8;
    }
}

int main(int argc,char *argv[]) {

    size_t nblocks = 4;
    size_t nthreads = 32;
    time_t startclock, endclock;
    float time;

    size_t n = nblocks * nthreads;
    size_t nbytes = 2;
    short ignore = 0;
    bits_t** bits_cpu;
    unsigned char* bits_gpu;
    unsigned long* debug_cpu;
    unsigned long* debug_gpu;

    // init cpu array
    bits_cpu = (bits_t**)calloc(n,sizeof(bits_t*));

    for(int i = 0; i < n; i ++) {
        bits_cpu[i] = bits_allocate(nbytes * 8 - ignore);
    }
    debug_cpu = (unsigned long*)calloc(n, sizeof(unsigned long));
    // init gpu array
    hipMalloc((void**)&bits_gpu,n*nbytes*sizeof(unsigned char));
    hipMalloc((void**)&debug_gpu, n*sizeof(unsigned long));

    // calling kernel
    generate_ids<<<nblocks, nthreads>>>(bits_gpu, nbytes, debug_gpu);
    hipDeviceSynchronize();
    // passing chars into bits_t and converting bits_t into unsigned long
    unsigned long* longs = (unsigned long*)calloc(n,sizeof(unsigned long));

    for(int i = 0; i < n; i ++) {
        // passing chars into bits_t
        hipMemcpy(bits_cpu[i]->byte,&bits_gpu[i*nbytes],nbytes*sizeof(unsigned char),hipMemcpyDeviceToHost);
        //converting bits_t into unsigned long
        longs[i] = bits_to_long(bits_cpu[i]);
    }
    hipMemcpy(debug_cpu, debug_gpu, n*sizeof(unsigned long), hipMemcpyDeviceToHost);

    // Verifying result
    // Visually

    for(int i = 0; i < n; i ++) {
        printf("%i, ",(int)debug_cpu[i]);
        printf("%i, ", (int)longs[i]);
        bits_println(bits_cpu[i]);
    }
    // Algo
    int result2 = 0;
    for(int i = 1; i < n; i ++) {
        if(longs[i - 1] != longs[i] - 1) result2 ++;
    }
    if(result2 > 0) printf("there is %i/%i problems", (int)result2, (int)n);
    else printf("There is no problems");
}