
/*
 * Conway's Game of Life
 *
 * basic sequential version
 *
 * AM
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <unistd.h>
#include <assert.h>
#include <time.h>

float compute_time(time_t start,time_t end)
{
   return ((float)((int)end - (int)start))/CLOCKS_PER_SEC;
};


typedef struct
{
   size_t n;
   size_t m;
   bool* t;
}  torus_t;

// converting (x,y) indices into one unique z index
size_t torus_index(size_t n,size_t m,size_t x,size_t y,int dx,int dy)
{
   assert(n > 0UL);
   assert(m > 0UL);
   long i = x + dx;
   long j = y + dy;
   if (i < 0)  i = n + i;
   if (j < 0)  j = m + j;
   i = i%n;
   j = j%m;
   return (size_t) i*m + j;  // row by row organization of cells
};

// torus memory allocation
torus_t* torus_allocate(size_t n,size_t m)
{
   assert(n > 0UL);
   assert(m > 0UL);
   torus_t *torus = NULL;
   torus = (torus_t*)malloc(sizeof(torus_t));
   if (torus == NULL)  return NULL;
   torus->n = n;
   torus->m = m;
   torus->t = NULL;
   torus->t = (bool*)calloc(n*m,sizeof(bool));
   if (torus->t == NULL)
   {
      free(torus);
      return NULL;
   };
   return torus;
};

// writing a torus cell location
void torus_write(torus_t *torus,size_t x,size_t y,bool value)
{
   size_t z = torus_index(torus->n,torus->m,x,y,0,0);
   torus->t[z] = value;
};

// reading a torus cell location
bool torus_read(torus_t *torus,size_t x,size_t y,int dx,int dy)
{
   size_t z = torus_index(torus->n,torus->m,x,y,dx,dy);
   return torus->t[z];
};

// random generating a torus (includes allocation)
torus_t* torus_random(size_t n,size_t m,double p)
{
   assert(n > 0UL);
   assert(m > 0UL);
   assert(p >= 0.0 && p <= 1.0);
   torus_t *torus = torus_allocate(n,m);
   for (int z = 0; z < n*m; z++)
   {
      bool cell = false;
      if (rand()%10000/10000.0 < p)  cell = true;
      torus->t[z] = cell;
   };
   return torus;
};

// loading a torus model from a file (includes allocation)
torus_t* torus_load(char* filename,bool DOSformat)
{
   // if your text file is encoded in DOS format,
   // two chars appear at the end of each line
   short endofline = 1;
   if (DOSformat)  endofline++;

   // attempting to open the file
   FILE *input = fopen(filename,"r");
   if (input == NULL)  return NULL;

   // estimating the torus size
   fseek(input,0,SEEK_END);
   size_t filesize = ftell(input);
   rewind(input);
   char line[256];
   fgets(line,sizeof(line),input);
   size_t m = ftell(input) - endofline;
   size_t n = filesize/m;
   torus_t *torus = torus_allocate(n,m);
   rewind(input);

   // reading the file char by char
   char c;
   size_t z = 0;
   while (EOF != (c = fgetc(input)))
   {
      if (c != '\n')
      {
         bool cell = false;
         if (c == 'x')  cell = true;
         torus->t[z] = cell;
         z++;
      };
   };

   // ending
   fclose(input);
   return torus;
};

// counting the number of alive neighbours for (x,y)-cell
size_t torus_neighbours(torus_t *torus,size_t x,size_t y)
{
   size_t count = 0;
   for (int dx = -1; dx <= 1; dx++)
   {
      for (int dy = -1; dy <= 1; dy++)
      {
         if (dx != 0 || dy != 0)
         {
            if (torus_read(torus,x,y,dx,dy))  count++;
         };
      };
   };
   return count;
};

// creating the next generation
void torus_next(torus_t *src,torus_t *dst)
{
   bool cell;
   size_t nn;
   for (size_t x = 0; x < src->n; x++)
   {
      for (size_t y = 0; y < src->m; y++)
      {
         cell = torus_read(src,x,y,0,0);
         nn = torus_neighbours(src,x,y);
         if (cell)  // alive
         {
            if (nn < 2 || nn > 3)  cell = false;  // dies
         }
         else  // dead
         {
            if (nn == 3)  cell = true;  // gets born
         };
         torus_write(dst,x,y,cell);
      };
   };
};

// printing the torus
void torus_print(torus_t* torus)
{
   for (int x = 0; x < torus->n; x++)
   {
      for (int y = 0; y < torus->m; y++)
      {
         bool cell = torus_read(torus,x,y,0,0);
         if (cell)  printf("x");  else  printf(" ");
      };
      printf("\n");
   };
};

// freeing the torus
void torus_free(torus_t *torus)
{
   free(torus->t);
   free(torus);
};

// drawing a separation line on the screen
void draw_line(size_t n)
{
   assert(n > 0UL);
   for (size_t i = 0; i < n; i++)  printf("-");
   printf("\n");
};

// converting (x,y) indices into one unique z index
size_t bool_index(size_t x,size_t y,int dx,int dy,size_t n,size_t m)
{
   assert(n > 0UL);
   assert(m > 0UL);
   long i = x + dx;
   long j = y + dy;
   if (i < 0)  i = n + i;
   if (j < 0)  j = m + j;
   i = i%n;
   j = j%m;
   return (size_t) i*m + j;  // row by row organization of cells
};

// reading a torus cell location
bool bool_read(bool *torus,size_t x,size_t y,int dx,int dy, size_t n, size_t m)
{
   size_t z = bool_index(x,y,dx,dy,n,m);
   return torus[z];
};

// printing the torus
void bool_print(bool* torus, size_t n, size_t m)
{
   for (int x = 0; x < n; x++)
   {
      for (int y = 0; y < m; y++)
      {
         bool cell = bool_read(torus,x,y,0,0,n,m);
         if (cell)  printf("x");  else  printf(" ");
      };
      printf("\n");
   };
};

__device__ size_t torus_index(size_t x,size_t y,int dx,int dy, size_t n, size_t m)
{
   long i = x + dx;
   long j = y + dy;
   if (i < 0)  i = n + i;
   if (j < 0)  j = m + j;
   i = i%n;
   j = j%m;
   return (size_t) i * m + j;  // row by row organization of cells
};


__device__ bool torus_read(bool *torus,size_t x, size_t y, int dx, int dy, size_t n, size_t m)
{
   size_t z = torus_index(x, y, dx, dy, n, m);
   return torus[z];
};


__device__ size_t torus_neighbours(bool* torus, size_t x, size_t y, size_t n, size_t m) {
    size_t count = 0;
    for (int dx = -1; dx <= 1; dx++)
    {
       for (int dy = -1; dy <= 1; dy++)
       {
          if (dx != 0 || dy != 0)
          {
             if (torus_read(torus, x, y, dx, dy, n, m))  count++;
          };
       };
    };
    return count;
}

__device__ void torus_write(bool *t, size_t x, size_t y, bool value, size_t n, size_t m)
{
   size_t z = torus_index(x,y,0,0,n,m);
   t[z] = value;
};


__device__ void cell_next(bool* src, bool* dst, size_t x, size_t y, size_t n, size_t m) {
    bool cell;
    size_t nn;
    cell = torus_read(src, x, y, 0, 0, n, m);
    nn = torus_neighbours(src, x, y, n, m);
    if (cell)  // alive
    {
        if (nn < 2 || nn > 3)  cell = false;  // dies
    }
    else  // dead
    {
        if (nn == 3)  cell = true;  // gets born
    }
    torus_write(dst, x, y, cell, n, m);
}

__global__ void run_step(bool* t_src, bool* t_dst) {
    size_t n = blockDim.x * gridDim.x;
    size_t m = blockDim.y * gridDim.y;
    size_t idx = (blockIdx.x*blockDim.x) + threadIdx.x;
    size_t idy = (blockIdx.y*blockDim.y) + threadIdx.y;
    cell_next(t_src, t_dst, idx, idy, n, m);

}

void compare_results(bool* cpu, bool* gpu, size_t n, size_t m) {
    size_t ndiff = 0;
    for(size_t i = 0; i < n; i ++) {
        for(size_t j = 0; j < m; j ++) {
            if(cpu[i *m + j] != gpu[i *m + j]) ndiff++;
        }
    }
    if(ndiff) printf("there is %i differences between gpu and cpu\n", (int)ndiff);
    else printf("there is no differences between gpu and cpu\n");
}


int main(int argc,char *argv[])
{
    size_t nstep = 10;
    bool gpu_running = true;
    bool cpu_running = true;
    bool print_grids = false;
    size_t nblocksx = 32;
    size_t nblocksy = 32;
    size_t nthreadsx = 64;
    size_t nthreadsy = 64;
    dim3 nblocks( nblocksx, nblocksy);
    dim3 nthreads( nthreadsx, nthreadsy);
    size_t nx = nblocksx * nthreadsx; // n
    size_t ny = nblocksy * nthreadsy; // m
    size_t n = nx * ny;
    time_t startclock, endclock;
    float time;
    
    
    printf("grid size : %i x %i, %i cells \n",(int)nx, (int)ny, (int)n);

    
    torus_t *t_src = torus_random(nx, ny, 0.1);
    torus_t *t_dst = torus_allocate(nx, ny);
    bool* b_src = (bool*)calloc(n,sizeof(bool));

    if(print_grids) {
        printf("printing initial config : \n");
        torus_print(t_src);
    }
    ////////////////////
    // GPU ////
    ////////////////////
    if(!gpu_running) goto cpu;
    bool* t_gpu_src;
    bool* t_gpu_dst;
    hipMalloc((void**)&t_gpu_src,n*sizeof(bool));
    hipMalloc((void**)&t_gpu_dst,n*sizeof(bool));
    
    hipMemcpy(t_gpu_src,t_src->t,n*sizeof(bool),hipMemcpyHostToDevice);
    printf("Computing game in gpu ... ");
    startclock = clock();
    for(size_t i = 0; i < nstep; i ++) {
        run_step<<<nblocks, nthreads>>>(t_gpu_src, t_gpu_dst);
        hipDeviceSynchronize();
        bool* tmp = t_gpu_src;
        t_gpu_src = t_gpu_dst;
        t_gpu_dst = tmp;
    }
    endclock = clock();
    time = compute_time(startclock,endclock);
    printf("done, in %7.5f s. \n", time);
    hipMemcpy(b_src,t_gpu_src,n*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    if(print_grids) {
        printf("printing gpu game : \n");
        bool_print(b_src, nx, ny);
    }


    
    ////////////////////
    // CPU ////
    ////////////////////
    cpu:
    if(!cpu_running) goto end;
    torus_t *tmp;
    printf("Computing game in cpu ... ");
    startclock = clock();
    for (size_t i = 0; i < nstep; i++)
    {
        torus_next(t_src,t_dst);
        //torus_print(t_dst);
        //draw_line(t_dst->m);
        //usleep(SPEED);
        tmp = t_src;
        t_src = t_dst;
        t_dst = tmp;
    };
    endclock = clock();
    time = compute_time(startclock,endclock);
    printf("done, in %7.5f s. \n", time);
    
    if(print_grids) {
        printf("printing cpu game : \n");
        torus_print(t_src);
    }
    end:
    if(gpu_running && cpu_running) compare_results(t_src->t, b_src, nx, ny);

    
}