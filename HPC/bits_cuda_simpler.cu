
/* bits type
 *
 * for simplicity, we only transfer the array of unsigned chars!
 *
 * AM
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <assert.h>

// bits_t type
struct bits
{
   size_t nbytes;
   short ignore;
   unsigned char *byte;
};
typedef struct bits bits_t;

// allocating memory for a bits_t instance
__host__ bits_t* bits_allocate(size_t nbits)
{
   bits_t *bits = NULL;
   bits = (bits_t*)malloc(sizeof(bits_t));
   if (bits == NULL)  return NULL;
   bits->nbytes = nbits/8;
   bits->ignore = 0;
   short remainder = nbits%8;
   if (remainder != 0)
   {
      bits->ignore = (short) (8 - remainder);
      bits->nbytes++;
   };
   bits->byte = NULL;
   bits->byte = (unsigned char*)calloc(bits->nbytes,sizeof(unsigned char));
   if (bits->byte == NULL)
   {
      free(bits);
      return NULL;
   }
   return bits;
};

// counting the number of actually used bits
__host__ size_t bits_actual_length(bits_t *bits)
{
   return 8*bits->nbytes - bits->ignore;
};

// printing a bits_t instance
__host__ void bits_print(bits_t *bits)
{
   // the list of bits
   printf("[");
   for (size_t k = 0; k < bits->nbytes; k++)
   {
      short b = 7;
      if (k == 0 && bits->ignore != 0)  b = b - bits->ignore;
      for (; b >= 0; b--)
      {
         printf("%1d",(bits->byte[k]>>b)&1);
      };
      if (k != bits->nbytes - 1)  printf("|");
   };
   printf("] ");

   // info about the bit string
   size_t actual_bits = bits_actual_length(bits);
   printf("(%lu;%lu)",actual_bits,bits->nbytes);
};

// printing a bits_t instance with new line
__host__ void bits_println(bits_t *bits)
{
   bits_print(bits);
   printf("\n");
};

// freeing the memory for a bits_t instance
__host__ void bits_free(bits_t *bits)
{
   free(bits->byte);
   free(bits);
};

// kernel for a random bit string generation with predefined number of zeros and ones
// -> the number of ones is set to the number of ones forming the thread id
__global__ void generate(size_t nbytes,unsigned char *array)
{
   size_t id = (blockIdx.x*blockDim.x) + threadIdx.x;
   size_t istart = id*nbytes;  // this thread's starting and ending elements in the array
   size_t iend = (id + 1)*nbytes;
   size_t k = 0;
   for (size_t i = istart; i < iend; i++)
   {
      array[i] = (unsigned char) (id >> k);
      k = k + 8;
   };
};


// main
int main(int argc,char *argv[])
{
   size_t nbits = 16;  // the length of the bit strings is fixed
   size_t warp_size = 32;
   size_t warps_per_block = 4;
   size_t nthreads = warp_size*warps_per_block;  // 128
   size_t nblocks = 512;
   size_t n = nblocks*nthreads;

   // welcome message
   fprintf(stderr,"Transfers of the bit strings stored in the bits_t types between RAM and GPU global memory\n");

   // memory allocation on the RAM
   fprintf(stderr,"Memory allocation on the RAM ... ");
   bits_t **cpu_side = (bits_t**)calloc(n,sizeof(bits_t*));
   for (size_t k = 0; k < n; k++)  cpu_side[k] = bits_allocate(nbits);
   fprintf(stderr,"done!\n");

   // allocation of one long array of unsigned chars on the global memory
   fprintf(stderr,"The memory allocation on the GPU ... ");
   unsigned char *gpu_side;
   hipMalloc((void**)&gpu_side,(n*nbits)*sizeof(unsigned char));
   fprintf(stderr,"done!\n");

   // invoking the kernel (each thread works on its piece of array)
   size_t nbytes = nbits/8;
   fprintf(stderr,"Invoking the kernel for generating the bit strings ...");
   generate<<<nblocks,nthreads>>>(nbytes,gpu_side);
   hipDeviceSynchronize();
   fprintf(stderr,"done!\n");

   // memory transfer back from GPU global memory
   fprintf(stderr,"Memory transfer of the 'unsigned char' array ... ");
   for (size_t k = 0; k < n; k++)  hipMemcpy(cpu_side[k]->byte,&gpu_side[k*nbytes],nbytes*sizeof(unsigned char),hipMemcpyDeviceToHost);
   fprintf(stderr,"done!\n");

   // visualization of the bits_t instances
   fprintf(stderr,"Visualization of the results ... ");
   for (size_t k = 0; k < n; k++)  bits_println(cpu_side[k]);

   // freeing memory
   for (size_t k = 0; k < n; k++)  bits_free(cpu_side[k]);
   free(cpu_side);
   hipFree(gpu_side);

   // ending
   fprintf(stderr,"We are done!\n");
   return 0;
};

