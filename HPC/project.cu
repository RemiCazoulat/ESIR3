// >>> Example of logs on a T4 gpu with 32 blocks in the grid and 64 threads per block >>>
/*
>>> Computing for the bits_t** array on gpu :
[bits_t a] init of gpu memory ... in 0.865703 s.
[bits_t a] calling first kernel ... in 0.000703 s.
[bits_t a] getting memory back on cpu ... in 1.648545 s.
first bits_t : 0 [00000000|00000000|00000000|00000000] (32;4)
...
last bits_t  : 65535 [00000000|00000000|11111111|11111111] (32;4)
[result] everything works well.

>>> Computing for the unsigned char* array on gpu :
[u char a] init of gpu memory ... in 0.000026 s.
[u char a] calling first kernel ... in 0.000082 s.
[u char a] getting memory back on cpu ... in 0.545371 s.
first bits_t : 0 [00000000|00000000|00000000|00000000] (32;4)
...
last bits_t  : 65535 [00000000|00000000|11111111|11111111] (32;4)
[result] everything works well.
 */
// >>>>>>> Interpretation >>>>>>>
// We can see on the logs that the init of the gpu memory is a lot longer with the bits_t** array than the char* array.
// It can be explained because the memory allocation is more complicated in the first case.

// Also, calling the kernel, even though it is doing the same operations, is longer with the bits_t** array.
// We can suppose that it is because the memory is more fragmented than the memory of a simple char* array, and accessing it could be longer.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <unistd.h>
#include <assert.h>
#include <time.h>

// >>>>>>> bits_t struct >>>>>>>
struct bits
{
    size_t nbytes;
    short ignore;
    unsigned char *byte;
};
typedef struct bits bits_t;

// >>>>>>> given functions >>>>>>>
__host__ float compute_time(time_t start,time_t end)
{
    return ((float)((int)end - (int)start))/CLOCKS_PER_SEC;
}

__host__ size_t bits_actual_length(bits_t *bits)
{
    return 8*bits->nbytes - bits->ignore;
}

__host__ void bits_print(bits_t *bits)
{
    // the list of bits
    printf("[");
    for (size_t k = 0; k < bits->nbytes; k++)
    {
        short b = 7;
        if (k == 0 && bits->ignore != 0)  b = b - bits->ignore;
        for (; b >= 0; b--)
        {
            printf("%1d",(bits->byte[k]>>b)&1);
        }
        if (k != bits->nbytes - 1)  printf("|");
    }
    printf("] ");

    // info about the bit string
    size_t actual_bits = bits_actual_length(bits);
    printf("(%zu;%zu)",actual_bits,bits->nbytes);
}

__host__ void bits_println(bits_t *bits)
{
    bits_print(bits);
    printf("\n");
}

__host__ bits_t* bits_allocate(size_t nbits)
{
    bits_t *bits = NULL;
    bits = (bits_t*)malloc(sizeof(bits_t));
    if (bits == NULL)  return NULL;
    bits->nbytes = nbits/8;
    bits->ignore = 0;
    short remainder = nbits%8;
    if (remainder != 0)
    {
        bits->ignore = (short) (8 - remainder);
        bits->nbytes++;
    };
    bits->byte = NULL;
    bits->byte = (unsigned char*)calloc(bits->nbytes,sizeof(unsigned char));
    if (bits->byte == NULL)
    {
        free(bits);
        return NULL;
    }
    return bits;
};

// >>>>>>> project functions >>>>>>>

// >>> Host functions >>>

// converts a bits_t into an unsigned long
__host__ unsigned long bits_to_long(bits_t* bits){
    unsigned long final_long = 0;
    for(int i = 1; i < bits->nbytes; i ++) {
        unsigned long current_long = (long)bits->byte[i] << (bits->nbytes - 1 - i) * 8;
        final_long += current_long;
    }
    // shifting last byte 2 times to get rid of ignored information
    unsigned char last_char = (unsigned char)bits->byte[0];
    last_char = (unsigned char)(last_char << bits->ignore);
    last_char = (unsigned char)(last_char >> bits->ignore);
    unsigned long last_long = (unsigned long)last_char << (bits->nbytes - 1) * 8;
    final_long += last_long;
    return final_long;
};

// takes the first n bits of an unsigned long, and converts it into a bits_t
__host__ bits_t* bits_from_long(size_t n, unsigned long value){
    if(n == 0) return NULL;
    bits_t* bits = bits_allocate(n);
    printf("[DEBUG] n_bytes : %zu\n", bits->nbytes);
    for(long i = (long)bits->nbytes - 1; i >= 0; i --) {
        char byte = (char)value;
        bits->byte[i] = byte;
        value = value >> 8;
    }
    return bits;
}

__host__ void print_results(unsigned long* longs, size_t n, bits_t** bits_cpu) {
    /*
    if (see_bytes) {
        for (int k = 0; k < n; k++) {
            printf("%lu ", longs[k]);
            bits_println(bits_cpu[k]);
        }
    }
    */

    printf("first bits_t : ");printf("%lu ", longs[0]); bits_println(bits_cpu[0]);
    printf("...\n");
    printf("last bits_t  : ");printf("%lu ", longs[n - 1]); bits_println(bits_cpu[n - 1]);

    int result2 = 0;
    for(int k = 1; k < n; k ++) {
        if(longs[k - 1] != longs[k] - 1) result2 ++;
    }
    if(result2 > 0) printf("[result] there is %d/%zu problems. \n", result2, n);
    else printf("[result] everything works well. \n");
};

// >>> kernels >>>
/**
 * Here, it is assumed that we no longer need the n, to avoid having 2 times the same bits_t.
 * the unsigned long value is now the id of the thread.
 * We only need an array of bits_t pointer to store the result.
 */
__global__ void bits_from_long_gpu(bits_t** bits) {
    unsigned long id = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned long value = id;
    for(long i = bits[id]->nbytes - 1; i >= 0; i --) {
        unsigned char byte = (unsigned char)value;
        bits[id]->byte[i] = byte;
        value = value >> 8;
    }
}

/**
 * Another kernel, that gets rid of the bits_t** array and replaces it by an unsigned char* array.
 * We this time need to have the nbytes argument, previsously stored in the bits_t struct.
 */
__global__ void chars_from_long_gpu(unsigned char* chars, size_t nbytes) {
    unsigned long id = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned long value = id;
    for(long i = nbytes - 1; i >= 0; i --) {
        unsigned char byte = (unsigned char)value;
        chars[i + id * nbytes] = byte;
        value = value >> 8;
    }
}

// >>>>>>> main >>>>>>>
int main(int argc,char *argv[]) {
    size_t nblocks = 32 * 32;
    size_t nthreads = 64;
    size_t n = nblocks * nthreads;
    size_t nbytes = 4; // or 32 bits
    short ignore = 0;

    bits_t** bits_cpu; // array for cpu
    bits_t** bits_gpu; // array for the first kernel
    unsigned char * chars_gpu; // array for the second kernel
    unsigned long* longs; // array to verify the kernels

    time_t startclock, endclock;
    float time;


    // >>>>>>> init cpu arrays >>>>>>>
    bits_cpu = (bits_t**)calloc(n,sizeof(bits_t*));
    longs = (unsigned long*)calloc(n,sizeof(unsigned long));

    // >>>>>>> --------------------------------- >>>>>>>
    //    >>>>>>> First try : bits_t** array on gpu >>>>>>>
    // >>>>>>> --------------------------------- >>>>>>>

    // >>>>>>> init gpu array >>>>>>>
    printf(">>> Computing for the bits_t** array on gpu : \n");
    printf("[bits_t a] init of gpu memory ...");
    startclock = clock();

    bits_t** tmp = (bits_t**)calloc(n, sizeof(bits_t*));
    for(int k = 0; k < n; k ++) {
        // creating temporary bits_t on cpu
        bits_t* bits_tmp = bits_allocate((nbytes * 8) - ignore);
        // freeing bits_tmp->byte now to avoid loosing the pointer and make impossible the free.
        free(bits_tmp->byte);
        // allocating a memory in gpu for byte array and putting the gpu pointer in bits_tmp
        hipMalloc((void**)&bits_tmp->byte, nbytes*sizeof(unsigned char));
        // allocating a memory space of the size of bits_t struct
        hipMalloc((void**)&tmp[k], sizeof(bits_t));
        // copying bits_tmp (that already contains the byte array on gpu) on gpu
        hipMemcpy(tmp[k], bits_tmp, sizeof(bits_t), hipMemcpyHostToDevice);
        // freeing memory on cpu
        free(bits_tmp);
    }
    hipMalloc((void**)&bits_gpu, n*sizeof(bits_t*));
    // copying every pointer of bits_t on gpu
    hipMemcpy(bits_gpu, tmp, n*sizeof(bits_t*), hipMemcpyHostToDevice);
    free(tmp);

    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> calling kernel >>>>>>>
    printf("[bits_t a] calling first kernel ...");
    startclock = clock();

    bits_from_long_gpu<<<nblocks, nthreads>>>(bits_gpu);
    hipDeviceSynchronize();

    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> getting memory back >>>>>>>
    printf("[bits_t a] getting memory back on cpu ...");
    startclock = clock();

    hipMemcpy(bits_cpu, bits_gpu, n*sizeof(bits_t*), hipMemcpyDeviceToHost);
    for(int k = 0; k < n; k ++) {
        // copying back the bits_t
        bits_t* bits_tmp = bits_allocate(nbytes * 8 - ignore);
        hipMemcpy(bits_tmp, bits_cpu[k], sizeof(bits_t), hipMemcpyDeviceToHost);
        hipFree(bits_cpu[k]); // free memory on gpu
        // copying back the byte array
        unsigned char* byte_tmp = (unsigned char*)malloc(nbytes * sizeof(unsigned char));
        hipMemcpy(byte_tmp, bits_tmp->byte, nbytes*sizeof(unsigned char),hipMemcpyDeviceToHost);
        hipFree(bits_tmp->byte); // free memory on gpu
        bits_tmp->byte = byte_tmp;
        bits_cpu[k] = bits_tmp;
    }

    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> transforming bits_t into long >>>>>>>
    for(int k = 0; k < n; k ++) {
        longs[k] = bits_to_long(bits_cpu[k]);
    }

    // >>>>>>> printing the result >>>>>>>
    print_results(longs, n, bits_cpu);

    // >>>>>>> --------------------------------- >>>>>>>
    //    >>>>>>> second try : unsigned char* array >>>>>>>
    // >>>>>>> --------------------------------- >>>>>>>

    // >>>>>>> init gpu array >>>>>>>
    printf("\n>>> Computing for the unsigned char* array on gpu : \n");
    printf("[u char a] init of gpu memory ...");
    startclock = clock();

    hipMalloc((void**)&chars_gpu,n*nbytes*sizeof(unsigned char));

    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> calling kernel >>>>>>>
    printf("[u char a] calling first kernel ...");
    startclock = clock();

    chars_from_long_gpu<<<nblocks, nthreads>>>(chars_gpu, nbytes);
    hipDeviceSynchronize();

    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> passing chars into bits_t >>>>>>>
    printf("[u char a] getting memory back on cpu ...");
    startclock = clock();
    for(int k = 0; k < n; k ++) {
        // passing chars into bits_t
        hipMemcpy(bits_cpu[k]->byte,&chars_gpu[k*nbytes],nbytes*sizeof(unsigned char),hipMemcpyDeviceToHost);
    }
    endclock = clock();
    time = compute_time(startclock, endclock);
    printf(" in %f s.\n", time);

    // >>>>>>> transforming bits_t into long >>>>>>>
    for(int k = 0; k < n; k ++) {
        longs[k] = bits_to_long(bits_cpu[k]);
    }

    // >>>>>>> printing the result >>>>>>>
    print_results(longs, n, bits_cpu);

    // >>>>>>> freeing memory >>>>>>>
    hipFree(bits_gpu);
    hipFree(chars_gpu);
    for(int k = 0; k < n; k ++) {
        free(bits_cpu[k]->byte);
        free(bits_cpu[k]);
    }
    free(bits_cpu);
    free(longs);

}